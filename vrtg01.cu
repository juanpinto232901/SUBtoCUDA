#include "hip/hip_runtime.h"
#include <Windows.h>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
//#include <hip/hip_vector_types.h>
//#include "BezierDefines.h"

#define MAX_RAND 2147483647
//#define USE_PBO
#define MAXCHARVAL 0.00392156862745
#define DEBUG_DEVICES

#define DEBUGVRaa

#define uint unsigned int
#define MAX_X 2048
#define MAX_Y 2048
#define MAX_Y2 2048 / 4


// 0-255 color values
//texture<uchar4, hipTextureType2D, hipReadModeElementType> textureIn;
// 0.0-1.0 color values
//texture<uchar4, hipTextureType2D,  hipReadModeNormalizedFloat> textureIn;
__constant__ float PI = 3.1415926535;



__device__ int randCu(uint* seed) // 1 <= *seed < m
{
    uint const a = 16807; //ie 7**5
    uint const m = MAX_RAND; //ie 2**31-1

    *seed = (*seed * a)%m;
    return(*seed);
}


__global__ void vrtogCU
(
        float *positions,
        char *colors,
        int *seedmem,
        float incMov
#ifdef DEBUGVR
        uint *debugmem0,
        uint *debugmem1,
        uint *debugmem2,
        float *debugmemFloat0,
        float *debugmemFloat1,
        float *debugmemFloat2,
        char *debugmemChar0,
        char *debugmemChar1,
        char *debugmemChar2
#endif
)
{
    unsigned int m = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int n = blockIdx.y * blockDim.y + threadIdx.y;

    if(m < MAX_X && n < MAX_Y)
    {
        uint iIndex = (n * MAX_X) + m;
        uint iIndex1= iIndex * 3;// * 3;
        uint iIndex2= iIndex * 4;
  //  // Park-Miller Pseudorandom Number Generator (PRNG).
        unsigned int seed = seedmem[m];
        float randNum1 = ((float)randCu(&seed) / (float)MAX_RAND);
        float randNum2 = ((float)randCu(&seed) / (float)MAX_RAND);
        float randNum3 = ((float)randCu(&seed) / (float)MAX_RAND);
        float randomZ1 = incMov * randNum1;// * 1.0; // Generate the random number.
//        float randomZ2 = incMov * randNum2 * 3.0;
//        float randomZ3 = incMov * randNum3 * 2.0;
        seedmem[m] = seed;
		positions[iIndex1 + 0] = (float)(m - (MAX_X * 0.5));// *0.25;// * 0.3;
		positions[iIndex1 + 1] = (float)(n - (MAX_Y * 0.5));// *0.25;// * 0.3;
		positions[iIndex1 + 2] = randomZ1;// *0.5;//(m * 0.3) + randomZ1;
        colors[iIndex2+0] = (char)(randNum1 * 255);
        colors[iIndex2+1] = (char)(randNum2 * 255);
        colors[iIndex2+2] = (char)(randNum3 * 255);
        colors[iIndex2+3] = (char)(255);
#ifdef DEBUGVR
        int iBase = 1 * 2048;
        if(iIndex >= iBase && iIndex < (iBase + 256)){
//blockIdx.x * blockDim.x + threadIdx.x
            debugmem0[m] = iIndex;
            debugmem1[m] = iIndex1;
            debugmem2[m] = iIndex2;
            debugmemFloat0[m] = positions[iIndex1+0];
            debugmemFloat1[m] = positions[iIndex1+1];
            debugmemFloat2[m] = positions[iIndex1+2];
            debugmemChar0[m] = colors[iIndex2+0];
            debugmemChar1[m] = colors[iIndex2+1];
            debugmemChar2[m] = colors[iIndex2+2];
        }
#endif
    }
//*/
}

__device__ int randCu01(uint* seed) // 1 <= *seed < m
{
	uint const a = 16807; //ie 7**5
	uint const m = MAX_RAND; //ie 2**31-1

	*seed = (*seed * a) % m;
	return(*seed);
}


__global__ void vrtogCU01
(
	float *positions,
	char *colors,
	int *seedmem,
	float incMov,
	uint iPosStart,
	uint iColStart
#ifdef DEBUGVR
	uint *debugmem0,
	uint *debugmem1,
	uint *debugmem2,
	float *debugmemFloat0,
	float *debugmemFloat1,
	float *debugmemFloat2,
	char *debugmemChar0,
	char *debugmemChar1,
	char *debugmemChar2
#endif
)
{
	int m = blockIdx.x * blockDim.x + threadIdx.x;
	int n = blockIdx.y * blockDim.y + threadIdx.y;

	if (m < MAX_X && n < MAX_Y2)
	{
		//uint iIndex = (n * MAX_X) + m;
		//uint iIndex1 = 0;// iPosStart + (iIndex * 3);
		//uint iIndex2 = 0;// iColStart + (iIndex * 4);
		////  // Park-Miller Pseudorandom Number Generator (PRNG).
		//unsigned int seed = 1.0;// seedmem[m];
		//float randNum1 = ((float)randCu01(&seed) / (float)MAX_RAND);
		//float randNum2 = ((float)randCu01(&seed) / (float)MAX_RAND);
		//float randNum3 = ((float)randCu01(&seed) / (float)MAX_RAND);
//		float randomZ1 = incMov * randNum1;// * 1.0; // Generate the random number.
//        float randomZ2 = incMov * randNum2 * 3.0;
//        float randomZ3 = incMov * randNum3 * 2.0;
//		seedmem[m] = seed;
		//positions[iIndex1 + 0] = (float)(m - (MAX_X * 0.5));// *0.25;// * 0.3;
		//positions[iIndex1 + 1] = (float)(n - (MAX_Y2 * 0.5));// *0.25;// * 0.3;
		//positions[iIndex1 + 2] = randomZ1;// *0.5;//(m * 0.3) + randomZ1;
		//colors[iIndex2 + 0] = (char)255;// (randNum1 * 255);
		//colors[iIndex2 + 1] = (char)(randNum2 * 255);
		//colors[iIndex2 + 2] = (char)(randNum3 * 255);
		//colors[iIndex2 + 3] = (char)(255);
#ifdef DEBUGVR
		int iBase = 1 * 2048;
		if (iIndex >= iBase && iIndex < (iBase + 256)) {
			//blockIdx.x * blockDim.x + threadIdx.x
			debugmem0[m] = iIndex;
			debugmem1[m] = iIndex1;
			debugmem2[m] = iIndex2;
			debugmemFloat0[m] = positions[iIndex1 + 0];
			debugmemFloat1[m] = positions[iIndex1 + 1];
			debugmemFloat2[m] = positions[iIndex1 + 2];
			debugmemChar0[m] = colors[iIndex2 + 0];
			debugmemChar1[m] = colors[iIndex2 + 1];
			debugmemChar2[m] = colors[iIndex2 + 2];
		}
#endif
	}
	//*/
}


__device__ int randCu02(uint* seed) // 1 <= *seed < m
{
	uint const a = 16807; //ie 7**5
	uint const m = MAX_RAND; //ie 2**31-1

	*seed = (*seed * a) % m;
	return(*seed);
}


__global__ void vrtogCU02
(
	float *positions,
	char *colors,
	int *seedmem,
	float incMov,
	uint iPosStart,
	uint iColStart
#ifdef DEBUGVR
	uint *debugmem0,
	uint *debugmem1,
	uint *debugmem2,
	float *debugmemFloat0,
	float *debugmemFloat1,
	float *debugmemFloat2,
	char *debugmemChar0,
	char *debugmemChar1,
	char *debugmemChar2
#endif
)
{
	unsigned int m = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int n = blockIdx.y * blockDim.y + threadIdx.y;

	if (m < MAX_X && n < MAX_Y2)
	{
		uint iIndex = (n * MAX_X) + m;
		uint iIndex1 = iPosStart + (iIndex * 3);
		uint iIndex2 = iColStart + (iIndex * 4);
		//  // Park-Miller Pseudorandom Number Generator (PRNG).
		unsigned int seed = seedmem[m];
		float randNum1 = ((float)randCu02(&seed) / (float)MAX_RAND);
		float randNum2 = ((float)randCu02(&seed) / (float)MAX_RAND);
		float randNum3 = ((float)randCu02(&seed) / (float)MAX_RAND);
		float randomZ1 = incMov * randNum1;// * 1.0; // Generate the random number.
//        float randomZ2 = incMov * randNum2 * 3.0;
//        float randomZ3 = incMov * randNum3 * 2.0;
		seedmem[m] = seed;
		positions[iIndex1 + 0] = (float)(m - (MAX_X * 0.5));// *0.25;// * 0.3;
		positions[iIndex1 + 1] = (float)(n - (MAX_Y2 * 0.5));// *0.25;// * 0.3;
		positions[iIndex1 + 2] = randomZ1;// *0.5;//(m * 0.3) + randomZ1;
		colors[iIndex2 + 0] = (char)(randNum1 * 255);
		colors[iIndex2 + 1] = (char)255;// (randNum2 * 255);
		colors[iIndex2 + 2] = (char)(randNum3 * 255);
		colors[iIndex2 + 3] = (char)(255);
#ifdef DEBUGVR
		int iBase = 1 * 2048;
		if (iIndex >= iBase && iIndex < (iBase + 256)) {
			//blockIdx.x * blockDim.x + threadIdx.x
			debugmem0[m] = iIndex;
			debugmem1[m] = iIndex1;
			debugmem2[m] = iIndex2;
			debugmemFloat0[m] = positions[iIndex1 + 0];
			debugmemFloat1[m] = positions[iIndex1 + 1];
			debugmemFloat2[m] = positions[iIndex1 + 2];
			debugmemChar0[m] = colors[iIndex2 + 0];
			debugmemChar1[m] = colors[iIndex2 + 1];
			debugmemChar2[m] = colors[iIndex2 + 2];
		}
#endif
	}
	//*/
}


__device__ int randCu03(uint* seed) // 1 <= *seed < m
{
	uint const a = 16807; //ie 7**5
	uint const m = MAX_RAND; //ie 2**31-1

	*seed = (*seed * a) % m;
	return(*seed);
}


__global__ void vrtogCU03
(
	float *positions,
	char *colors,
	int *seedmem,
	float incMov,
	uint iPosStart,
	uint iColStart
#ifdef DEBUGVR
	uint *debugmem0,
	uint *debugmem1,
	uint *debugmem2,
	float *debugmemFloat0,
	float *debugmemFloat1,
	float *debugmemFloat2,
	char *debugmemChar0,
	char *debugmemChar1,
	char *debugmemChar2
#endif
)
{
	unsigned int m = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int n = blockIdx.y * blockDim.y + threadIdx.y;

	if (m < MAX_X && n < MAX_Y2)
	{
		uint iIndex = (n * MAX_X) + m;
		uint iIndex1 = iPosStart + (iIndex * 3);
		uint iIndex2 = iColStart + (iIndex * 4);
		//  // Park-Miller Pseudorandom Number Generator (PRNG).
		unsigned int seed = seedmem[m];
		float randNum1 = ((float)randCu03(&seed) / (float)MAX_RAND);
		float randNum2 = ((float)randCu03(&seed) / (float)MAX_RAND);
		float randNum3 = ((float)randCu03(&seed) / (float)MAX_RAND);
		float randomZ1 = incMov * randNum1;// * 1.0; // Generate the random number.
//        float randomZ2 = incMov * randNum2 * 3.0;
//        float randomZ3 = incMov * randNum3 * 2.0;
		seedmem[m] = seed;
		positions[iIndex1 + 0] = (float)(m - (MAX_X * 0.5));// *0.25;// * 0.3;
		positions[iIndex1 + 1] = (float)(n - (MAX_Y2 * 0.5));// *0.25;// * 0.3;
		positions[iIndex1 + 2] = randomZ1;// *0.5;//(m * 0.3) + randomZ1;
		colors[iIndex2 + 0] = (char)(randNum1 * 255);
		colors[iIndex2 + 1] = (char)(randNum2 * 255);
		colors[iIndex2 + 2] = (char)255;// (randNum3 * 255);
		colors[iIndex2 + 3] = (char)(255);
#ifdef DEBUGVR
		int iBase = 1 * 2048;
		if (iIndex >= iBase && iIndex < (iBase + 256)) {
			//blockIdx.x * blockDim.x + threadIdx.x
			debugmem0[m] = iIndex;
			debugmem1[m] = iIndex1;
			debugmem2[m] = iIndex2;
			debugmemFloat0[m] = positions[iIndex1 + 0];
			debugmemFloat1[m] = positions[iIndex1 + 1];
			debugmemFloat2[m] = positions[iIndex1 + 2];
			debugmemChar0[m] = colors[iIndex2 + 0];
			debugmemChar1[m] = colors[iIndex2 + 1];
			debugmemChar2[m] = colors[iIndex2 + 2];
		}
#endif
	}
	//*/
}


__device__ int randCu04(uint* seed) // 1 <= *seed < m
{
	uint const a = 16807; //ie 7**5
	uint const m = MAX_RAND; //ie 2**31-1

	*seed = (*seed * a) % m;
	return(*seed);
}


__global__ void vrtogCU04
(
	float *positions,
	char *colors,
	int *seedmem,
	float incMov,
	uint iPosStart,
	uint iColStart
#ifdef DEBUGVR
	uint *debugmem0,
	uint *debugmem1,
	uint *debugmem2,
	float *debugmemFloat0,
	float *debugmemFloat1,
	float *debugmemFloat2,
	char *debugmemChar0,
	char *debugmemChar1,
	char *debugmemChar2
#endif
)
{
	unsigned int m = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int n = blockIdx.y * blockDim.y + threadIdx.y;

	if (m < MAX_X && n < MAX_Y2)
	{
		uint iIndex = (n * MAX_X) + m;
		uint iIndex1 = iPosStart + (iIndex * 3);
		uint iIndex2 = iColStart + (iIndex * 4);
		//  // Park-Miller Pseudorandom Number Generator (PRNG).
		unsigned int seed = seedmem[m];
		float randNum1 = ((float)randCu04(&seed) / (float)MAX_RAND);
		float randNum2 = ((float)randCu04(&seed) / (float)MAX_RAND);
		float randNum3 = ((float)randCu04(&seed) / (float)MAX_RAND);
		float randomZ1 = incMov * randNum1;// * 1.0; // Generate the random number.
//        float randomZ2 = incMov * randNum2 * 3.0;
//        float randomZ3 = incMov * randNum3 * 2.0;
		seedmem[m] = seed;
		positions[iIndex1 + 0] = (float)(m - (MAX_X * 0.5));// *0.25;// * 0.3;
		positions[iIndex1 + 1] = (float)(n - (MAX_Y2 * 0.5));// *0.25;// * 0.3;
		positions[iIndex1 + 2] = randomZ1;// *0.5;//(m * 0.3) + randomZ1;
		colors[iIndex2 + 0] = (char)(randNum1 * 255);
		colors[iIndex2 + 1] = (char)128 + (randNum2 * 128);
		colors[iIndex2 + 2] = (char)255;// (randNum3 * 255);
		colors[iIndex2 + 3] = (char)(255);
#ifdef DEBUGVR
		int iBase = 1 * 2048;
		if (iIndex >= iBase && iIndex < (iBase + 256)) {
			//blockIdx.x * blockDim.x + threadIdx.x
			debugmem0[m] = iIndex;
			debugmem1[m] = iIndex1;
			debugmem2[m] = iIndex2;
			debugmemFloat0[m] = positions[iIndex1 + 0];
			debugmemFloat1[m] = positions[iIndex1 + 1];
			debugmemFloat2[m] = positions[iIndex1 + 2];
			debugmemChar0[m] = colors[iIndex2 + 0];
			debugmemChar1[m] = colors[iIndex2 + 1];
			debugmemChar2[m] = colors[iIndex2 + 2];
		}
#endif
	}
	//*/
}

